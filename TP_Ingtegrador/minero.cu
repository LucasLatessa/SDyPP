#include "hip/hip_runtime.h"
#include <stdint.h>
#include <string.h>
#include <stdlib.h>
#include <stdio.h>
#include "cuda_md5.cu"

__device__ bool starts_with(const uint8_t* hash, const uint8_t* prefix, int prefix_len) {
	for (int i = 0; i < prefix_len; ++i) {
		if (hash[i] != prefix[i])
			return false;
	}
	
	return true;
}

__global__ void cuda_minero(const uint8_t* buffer, size_t buffer_len, int from, const uint8_t* prefix, int prefix_len, int* nonce) {
	int _nonce = from + blockIdx.x * blockDim.x + threadIdx.x;
	
	uint8_t local_buffer[40];
	uint8_t result[32];

	uint32_to_hex(local_buffer, _nonce, 0);
	for ( int i = 8; i < 40; i++ ) {
		local_buffer[i] = buffer[i - 8];
	}
		
	cuda_md5(local_buffer, 40, result);
	
	//if ( (*nonce == 0) && starts_with(result, prefix, prefix_len) ) {
	//	printf("Voy a setear %d / %d\n", _nonce, *nonce);		
	//}
	//
	//if ( (*nonce != 0) && starts_with(result, prefix, prefix_len) ) {
	//	printf("Alguien me gano %d / %d\n", _nonce, *nonce);		
	//}
	//
	//*nonce = ( (*nonce == 0) && starts_with(result, prefix, prefix_len) ? _nonce : 0 );
	
	int i = 0;
	while ( *nonce == 0 && i <= _nonce ) {
		if ( (i == _nonce) && starts_with(result, prefix, prefix_len) ) {
			printf("Setea resultado %d / %d\n", _nonce, *nonce);
			*nonce = _nonce;
		}
		
		i++;
	}
}

int main(int argc, char *argv[]) {
	if (argc != 5) {
    printf("Use: miner FROM TO PREFIX HASH\n");
    printf("Where: \nFROM: integer\nTO: integer\nPREFIX: string\nHASH: string[32]\n");
		return 0;		
	}
		
	int from = atoi(argv[1]);
	int to = atoi(argv[2]);
	
	const char* prefix = argv[3];
	const char* buffer = argv[4];
	
	size_t buffer_len = strlen(buffer);
	size_t prefix_len = strlen(prefix);

	uint8_t* dev_buffer;
	uint8_t* dev_prefix;
	int* dev_nonce;

	hipMalloc((void**)&dev_buffer, (buffer_len + sizeof(int)) * sizeof(uint8_t));
	hipMalloc((void**)&dev_prefix, prefix_len * sizeof(uint8_t));
	hipMalloc((void**)&dev_nonce, sizeof(int));

	hipMemcpy(dev_buffer, buffer, buffer_len * sizeof(uint8_t), hipMemcpyHostToDevice);
	hipMemcpy(dev_prefix, prefix, prefix_len * sizeof(uint8_t), hipMemcpyHostToDevice);
	//hipMemcpy(dev_nonce, nonce, sizeof(int), hipMemcpyHostToDevice);
	
	int threads = 512;
	int blocks  = round((to - from) / threads);
	/*
		There are multiple limits. 
		All must be satisfied.
	
		The maximum number of threads in the block is limited to 1024. 
		This is the product of whatever your threadblock dimensions are (xyz). 
		For example (32,32,1) creates a block of 1024 threads. (33,32,1) is not legal, since 33*32*1 > 1024.
	
		The maximum x-dimension is 1024. (1024,1,1) is legal. (1025,1,1) is not legal.
		The maximum y-dimension is 1024. (1,1024,1) is legal. (1,1025,1) is not legal.
		The maximum z-dimension is 64. (1,1,64) is legal. (2,2,64) is also legal. (1,1,65) is not legal.
	
		Also, threadblock dimensions of 0 in any position are not legal.
		Your choice of threadblock dimensions (x,y,z) must satisfy each of the rules 1-4 above.
	*/

	cuda_minero<<<blocks, threads>>>(dev_buffer, buffer_len, from, dev_prefix, prefix_len, dev_nonce);
	hipDeviceSynchronize();
	
	hipError_t error = hipGetLastError();
	
	if (error != hipSuccess) {
		printf("{ \"error\": true, \"cuda\": \"%s\" }", hipGetErrorString(error));
		return 1;
	}
	
	int nonce = 0;
	hipMemcpy(&nonce, dev_nonce, sizeof(int), hipMemcpyDeviceToHost);
	
	hipFree(dev_buffer);
	hipFree(dev_prefix);
	hipFree(dev_nonce);
	
	if ( nonce > 0 ) {
		printf("{ \"error\": false, \"nonce\": %d, \"hex\": \"%08x\" }", nonce, nonce);
	} else {
		printf("{ \"error\": true, \"from\": %d, \"to\": %d, \"prefix\": \"%s\", \"hash\": \"%s\" }", from, to, prefix, buffer);
	}

	return 0;
}