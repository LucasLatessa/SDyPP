
#include <hip/hip_runtime.h>
/**
 **********************************************************************
 ** Copyright (C) 1990, RSA Data Security, Inc. All rights reserved. **
 **                                                                  **
 ** License to copy and use this software is granted provided that   **
 ** it is identified as the "RSA Data Security, Inc. MD5 Message     **
 ** Digest Algorithm" in all material mentioning or referencing this **
 ** software or this function.                                       **
 **                                                                  **
 ** License is also granted to make and use derivative works         **
 ** provided that such works are identified as "derived from the RSA **
 ** Data Security, Inc. MD5 Message Digest Algorithm" in all         **
 ** material mentioning or referencing the derived work.             **
 **                                                                  **
 ** RSA Data Security, Inc. makes no representations concerning      **
 ** either the merchantability of this software or the suitability   **
 ** of this software for any particular purpose.  It is provided "as **
 ** is" without express or implied warranty of any kind.             **
 **                                                                  **
 ** These notices must be retained in any copies of any part of this **
 ** documentation and/or software.                                   **
 **********************************************************************
 */

__device__ uint32_t shifts[] = {  7, 12, 17, 22,
                                  5,  9, 14, 20,
																	4, 11, 16, 23,
																	6, 10, 15, 21 };

__device__ uint32_t sines[]  = { 0xd76aa478, 0xe8c7b756, 0x242070db, 0xc1bdceee,
																 0xf57c0faf, 0x4787c62a, 0xa8304613, 0xfd469501,
																 0x698098d8, 0x8b44f7af, 0xffff5bb1, 0x895cd7be,
																 0x6b901122, 0xfd987193, 0xa679438e, 0x49b40821,
																 0xf61e2562, 0xc040b340, 0x265e5a51, 0xe9b6c7aa,
																 0xd62f105d, 0x02441453, 0xd8a1e681, 0xe7d3fbc8,
																 0x21e1cde6, 0xc33707d6, 0xf4d50d87, 0x455a14ed,
																 0xa9e3e905, 0xfcefa3f8, 0x676f02d9, 0x8d2a4c8a,
																 0xfffa3942, 0x8771f681, 0x6d9d6122, 0xfde5380c,
																 0xa4beea44, 0x4bdecfa9, 0xf6bb4b60, 0xbebfbc70,
																 0x289b7ec6, 0xeaa127fa, 0xd4ef3085, 0x04881d05,
																 0xd9d4d039, 0xe6db99e5, 0x1fa27cf8, 0xc4ac5665,
																 0xf4292244, 0x432aff97, 0xab9423a7, 0xfc93a039,
																 0x655b59c3, 0x8f0ccc92, 0xffeff47d, 0x85845dd1,
																 0x6fa87e4f, 0xfe2ce6e0, 0xa3014314, 0x4e0811a1,
																 0xf7537e82, 0xbd3af235, 0x2ad7d2bb, 0xeb86d391 };

__device__ uint32_t left_rotate(uint32_t x, uint32_t c) {
	return ((x << c) | (x >> (32 - c)));
}

__device__ void uint32_to_hex(uint8_t *str, uint32_t num, int skip) {
	const char *hex_digits = "0123456789abcdef";

	for ( int i = 0; i < 8; i++ ) {
		str[skip + i] = hex_digits[(num >> (28 - 4 * i)) & 0xF];
	}
}

__device__ void cuda_md5(const uint8_t* buffer, size_t buffer_len, uint8_t* result) {
	int blocks = (buffer_len + 8) / 64 + 1;

	uint32_t aa = 0x67452301;
	uint32_t bb = 0xefcdab89;
	uint32_t cc = 0x98badcfe;
	uint32_t dd = 0x10325476;

	for (int i = 0; i < blocks; i++) {
		const uint8_t* block = buffer;
		int offset = i * 64;

		if (offset + 64 > buffer_len) {
			uint8_t* padded_block = (uint8_t*)malloc(64);

			for (int j = offset; j < buffer_len; j++) {
					padded_block[j - offset] = buffer[j];
			}
			
			if (offset <= buffer_len) {
					padded_block[buffer_len - offset] = 0x80;
			}
			
			if (i == blocks - 1) {
					padded_block[56] = (uint8_t)(buffer_len << 3);
					padded_block[57] = (uint8_t)(buffer_len >> 5);
					padded_block[58] = (uint8_t)(buffer_len >> 13);
					padded_block[59] = (uint8_t)(buffer_len >> 21);
			}

			block = padded_block;
			offset = 0;
		}

		uint32_t a = aa;
		uint32_t b = bb;
		uint32_t c = cc;
		uint32_t d = dd;

		uint32_t f;
		int g;

		for (int j = 0; j < 64; j++) {
			if (j < 16) {
					f = (b & c) | (~b & d);
					g = j;
			}
			else if (j < 32) {
					f = (b & d) | (c & ~d);
					g = 5 * j + 1;
			}
			else if (j < 48) {
					f = b ^ c ^ d;
					g = 3 * j + 5;
			}
			else {
					f = c ^ (b | ~d);
					g = 7 * j;
			}

			g = (g & 0x0f) * 4 + offset;

			uint32_t hold = d;
			d = c;
			c = b;

			b = a + f + sines[j] + (uint32_t)(block[g] + (block[g + 1] << 8) + (block[g + 2] << 16) + (block[g + 3] << 24));
			b = b << shifts[j & 3 | j >> 2 & ~3] | b >> 32 - shifts[j & 3 | j >> 2 & ~3];
			b += c;

			a = hold;
		}

		aa += a;
		bb += b;
		cc += c;
		dd += d;

		if (offset != 0)
			free((void*)block);
	}
	
	uint32_to_hex(result, aa,  0);
	uint32_to_hex(result, bb,  8);
	uint32_to_hex(result, cc, 16);
	uint32_to_hex(result, dd, 24);
}