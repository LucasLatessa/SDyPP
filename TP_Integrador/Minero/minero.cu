#include "hip/hip_runtime.h"
#include <stdint.h>
#include <string.h>
#include <stdlib.h>
#include <stdio.h>
#include "cuda_md5.cu"

__device__ bool starts_with(const uint8_t* hash, const uint8_t* prefix, int prefix_len) {
	for (int i = 0; i < prefix_len; ++i) {
		if (hash[i] != prefix[i])
			return false;
	}
	return true;
}

__global__ void cuda_minero(const uint8_t* buffer, size_t buffer_len, int from, const uint8_t* prefix, int prefix_len, int* nonce) {
	int _nonce = from + blockIdx.x * blockDim.x + threadIdx.x;
	
	uint8_t local_buffer[40];
	uint8_t result[32];

	// Prepare the buffer with the nonce and data
	uint32_to_hex(local_buffer, _nonce, 0);
	for (int i = 8; i < 40; i++) {
		local_buffer[i] = buffer[i - 8];
	}
		
	// Compute the MD5 hash
	cuda_md5(local_buffer, 40, result);
	
	// Check if the hash starts with the given prefix
	if (starts_with(result, prefix, prefix_len)) {
		// Attempt to set the nonce if it hasn't been set yet
		atomicCAS(nonce, 0, _nonce);
	}
}

int main(int argc, char *argv[]) {
	if (argc != 5) {
		printf("Use: miner FROM TO PREFIX HASH\n");
		printf("Where: \nFROM: integer\nTO: integer\nPREFIX: string\nHASH: string[32]\n");
		return 0;
	}
		
	int from = atoi(argv[1]);
	int to = atoi(argv[2]);
	
	const char* prefix = argv[3];
	const char* buffer = argv[4];
	
	size_t buffer_len = strlen(buffer);
	size_t prefix_len = strlen(prefix);

	uint8_t* dev_buffer;
	uint8_t* dev_prefix;
	int* dev_nonce;

	hipMalloc((void**)&dev_buffer, (buffer_len + sizeof(int)) * sizeof(uint8_t));
	hipMalloc((void**)&dev_prefix, prefix_len * sizeof(uint8_t));
	hipMalloc((void**)&dev_nonce, sizeof(int));

	hipMemcpy(dev_buffer, buffer, buffer_len * sizeof(uint8_t), hipMemcpyHostToDevice);
	hipMemcpy(dev_prefix, prefix, prefix_len * sizeof(uint8_t), hipMemcpyHostToDevice);

	// Initialize nonce to 0 on the device
	int nonce = 0;
	hipMemcpy(dev_nonce, &nonce, sizeof(int), hipMemcpyHostToDevice);
	
	int threads = 512;
	int blocks  = (to - from + threads - 1) / threads; // Ensure all threads are covered

	// Launch the CUDA kernel
	cuda_minero<<<blocks, threads>>>(dev_buffer, buffer_len, from, dev_prefix, prefix_len, dev_nonce);
	hipDeviceSynchronize();
	
	hipError_t error = hipGetLastError();
	
	if (error != hipSuccess) {
		printf("{ \"error\": true, \"cuda\": \"%s\" }", hipGetErrorString(error));
		return 1;
	}
	
	// Copy the nonce result back to the host
	hipMemcpy(&nonce, dev_nonce, sizeof(int), hipMemcpyDeviceToHost);
	
	// Free device memory
	hipFree(dev_buffer);
	hipFree(dev_prefix);
	hipFree(dev_nonce);
	
	if (nonce > 0) {
		printf("{ \"error\": false, \"nonce\": %d, \"hex\": \"%08x\" }", nonce, nonce);
	} else {
		printf("{ \"error\": true, \"from\": %d, \"to\": %d, \"prefix\": \"%s\", \"hash\": \"%s\" }", from, to, prefix, buffer);
	}

	return 0;
}
