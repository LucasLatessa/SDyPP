
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloWorld()
{
    printf("Hola Mundo desde la GPU!\n");
}

int main()
{
    helloWorld<<<1, 1>>>();

    hipDeviceSynchronize();

    return 0;
}
