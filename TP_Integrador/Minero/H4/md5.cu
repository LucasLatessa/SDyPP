#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "cuda_md5.cu"

__global__
void calculate_md5(const unsigned char* input, unsigned long long input_len, unsigned char* result) {
    cuda_md5(input, input_len, result); //calcula el hash con md5
}

int main(int argc, char *argv[]) {
    if (argc != 2) { //si no se pasa la cadena a hasher, muesta lo siguiente
        fprintf(stderr, "Uso: %s <cadena>\n", argv[0]);
        return 1;
    }

    const char* input = argv[1]; //agarra la palabra a hashear
    size_t input_len = strlen(input); // calcula la longutud

    unsigned char result[16]; // array de 16 bytes

    unsigned char* d_input; // asigna de la cadena puntero
    unsigned char* d_result; // asigna de respuesta puntero
    hipMalloc(&d_input, input_len * sizeof(unsigned char)); // asigna puntero de la cadena en gpu
    hipMalloc(&d_result, 16 * sizeof(unsigned char)); // asigna puntero de respuesta en gpu
    hipMemcpy(d_input, reinterpret_cast<const unsigned char*>(input), input_len * sizeof(unsigned char), hipMemcpyHostToDevice); // copia cadena desde cpu a gpu

    calculate_md5<<<1, 1>>>(d_input, input_len, d_result); // llama a la funcion del kernel

    hipMemcpy(result, d_result, 16 * sizeof(unsigned char), hipMemcpyDeviceToHost); //copia el resultado desde gpu a cpu

    printf("Hash MD5 de '%s': ", input); // muestra el resultado
    for (int i = 0; i < 16; ++i) {
        printf("%02x", result[i]);
    }
    printf("\n");

    hipFree(d_input); // libera memoria
    hipFree(d_result);

    return 0;
}
