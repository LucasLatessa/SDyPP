#include "hip/hip_runtime.h"
    // main.cu

    #include <stdio.h>
    #include <stdlib.h>
    #include <string.h>
    #include "cuda_md5.cu"

    __global__
    void calculate_md5(const unsigned char* input, unsigned long long input_len, unsigned char* result) {
        cuda_md5(input, input_len, result);
        printf("%s", result);
    }

    void byte_to_hex(const unsigned char* byte_array, char* hex_string, size_t length) {
        const char hex_digits[] = "0123456789abcdef";
        for (size_t i = 0; i < length; ++i) {
            hex_string[i * 2] = hex_digits[(byte_array[i] >> 4) & 0x0F];
            hex_string[i * 2 + 1] = hex_digits[byte_array[i] & 0x0F];
        }
        hex_string[length * 2] = '\0'; // Null-terminate the string
    }

    int main(int argc, char *argv[]) {
        if (argc != 2) {
            fprintf(stderr, "Uso: %s <cadena>\n", argv[0]);
            return 1;
        }

        const char* input = argv[1];
        size_t input_len = strlen(input);

        unsigned char result[16]; // MD5 produce un hash de 16 bytes

        unsigned char* d_input;
        unsigned char* d_result;
        hipMalloc(&d_input, input_len * sizeof(unsigned char));
        hipMalloc(&d_result, 16 * sizeof(unsigned char));
        hipMemcpy(d_input, reinterpret_cast<const unsigned char*>(input), input_len * sizeof(unsigned char), hipMemcpyHostToDevice);

        calculate_md5<<<1, 1>>>(d_input, input_len, d_result);

        hipMemcpy(result, d_result, 16 * sizeof(unsigned char), hipMemcpyDeviceToHost);

        char hex_result[33]; // 32 hex characters + null terminator
        byte_to_hex(result, hex_result, 16);

        printf("Hash MD5 de '%s': %s\n", input, hex_result);

        hipFree(d_input);
        hipFree(d_result);

        return 0;
    }
